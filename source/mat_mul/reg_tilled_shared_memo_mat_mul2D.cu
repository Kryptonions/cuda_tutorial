#include "hip/hip_runtime.h"
#include <iostream>
#include "../tools/common.cuh"

using namespace std;

#define V 8  // 每个thread负责计算的元素数量V*V
#define TILE_WIDTH 16
#define S TILE_WIDTH*V
#define L TILE_WIDTH*V

struct Matrix
{
    int width;
    int height;
    float *elements;
};

// 获取矩阵A的(row, col)元素
__device__ float getElement(Matrix *A, int row, int col)
{
    return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

// 矩阵相乘kernel，2-D，每个线程计算V*V大小的block
__global__ void regTilledSharedMemoryMatMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
    // block内共享内存，矩阵维度必须是常量，否则编译报错
    __shared__ float sharedA[L][S];
    __shared__ float sharedB[S][L];

    //__shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];
    float c[V][V] = {0};
    float a[V], b[V];

	int row = threadIdx.y + blockIdx.y * blockDim.y;
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

    //int m = A->height;                //A的行数
    int n = A->width;                   //A的列数
    //int k = B->width;                 //B的列数
    for (int i = 0; i < n; i += S) {
        // global memory to shared memory
        int aBaseRow = row * V;
        int aBaseCol = i * S + tx * V;
        int bBaseRow = i * S + ty * V;
        int bBaseCol = col * V;
        for (int j = 0; j < V; j++) {
            for (int k = 0; k < V; k++) {
                sharedA[ty * V + j][tx * V + k] = getElement(A, aBaseRow + j, aBaseCol + k);
                sharedB[ty * V + j][tx * V + k] = getElement(B, bBaseRow + j, bBaseCol + k);
            }
        }
        __syncthreads();
        for (int j = 0; j < S; j++) {
            // shared memory to register
            for (int k = 0; k < V; k++) {
                a[k] = sharedA[ty * V + k][j];
            }
            for (int k = 0; k < V; k++) {
                b[k] = sharedB[j][tx * V + k];
            }
            for (int j = 0; j < V; j++) {
                for (int k = 0; k < V; k++) {
                    c[j][k] += a[j] * b[k];
                }
            }
        }
    }
    // copy back
    for (int j = 0; j < V; j++) {
        for (int k = 0; k < V; k++) {
            setElement(C, row * V + j, col * V + k, c[j][k]);
        }
    }
}

int main(void)
{
    // 1、设置GPU设备
    setGPU();

    // 2、分配主机内存和设备内存，并初始化
    int width = 1 << 10;
    int height = 1 << 10;


    Matrix *A, *B, *C;
    // 申请托管内存
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }


    // 定义kernel的执行配置
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    // 每个线程负责V*V个值的计算
    dim3 gridSize((width + blockSize.x * V - 1) / (blockSize.x * V),
        (height + blockSize.y * V - 1) / (blockSize.y * V));
    // 执行kernel
    cout << (width + blockSize.x * V - 1) / (blockSize.x * V) << endl;
    cout << (height + blockSize.y * V - 1) / (blockSize.y * V) << endl;
    regTilledSharedMemoryMatMulKernel <<< gridSize, blockSize >>>(A, B, C);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < width * height; ++i)
        maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
    cout << "最大误差: " << maxError << endl;

    return 0;
}