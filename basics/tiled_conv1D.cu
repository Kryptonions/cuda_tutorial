#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cstdlib>
#include "../tools/common.cuh"

#define MASK_WIDTH 5
#define RADIUS MASK_WIDTH / 2
// 每个tile需要写的数量
#define O_TILE_WIDTH 10 
// 每个tile需要读的数量，需要额外读取更多数据，所以block内的线程数多于每个block需要写的量，一部分线程只负责读数据
#define BLOCK_WIDTH (O_TILE_WIDTH + MASK_WIDTH - 1) 

// mask数据在常量内存里
__constant__ int M[MASK_WIDTH];


__global__ void tiled1DConvKernel(int *N, int *P, int width) {
    // 共享内存
    __shared__ int Ns[BLOCK_WIDTH];  

    // 当前线程负责写的位置
    int index_o = blockIdx.x * O_TILE_WIDTH + threadIdx.x;
    // 当前线程负责读的位置
    int index_i = index_o - RADIUS;
    int tx = threadIdx.x;
    int output = 0;
    if ((index_i >= 0) && (index_i < width)) {
        Ns[tx] = N[index_i];
    } else {
        Ns[tx] = 0;
    }
    //等待所有线程load数据到共享内存
    __syncthreads();
    //只有前O_TILE_WIDTH个线程负责写数据，其余空闲
    if (tx < O_TILE_WIDTH) {
        output = 0;
        for (int j = 0; j < MASK_WIDTH; j++) {
            output += M[j] * Ns[j + tx];
        }
        P[index_o] = output;
    }
}

void verify_result(int *array, int *result, int *mask, int n) {
    // pad array with 0s
    int r = MASK_WIDTH / 2;
    int n_p = n + r * 2;
    int *h_array = new int[n_p];
    for (int i = 0; i < n_p; i++) {
        if ((i < r) || (i >= (n + r))) {
            h_array[i] = 0;
        } else {
            h_array[i] = array[i - r];
        }
    }
    int temp;
    for (int i = 0; i < n; i++) {
        temp = 0;
        for (int j = 0; j < MASK_WIDTH; j++) {
            temp += h_array[i + j] * mask[j];
        }
        //if (i < 20) {
        //  printf("temp=%d, result=%d\n", temp, result[i]);
        //}
        //if (temp != result[i]) {
        //printf("i=%d, temp=%d, result=%d\n", i, temp, result[i]);
        //}
        assert(temp == result[i]);
    }
}

int main(void) {
    setGPU();

    // 数组长度
    int width = 1 << 20;

    int *A, *P;
    int nBytes = width * sizeof(int);
    int mBytes = MASK_WIDTH * sizeof(int);

    // 内存托管
    hipMallocManaged((void**)&A, nBytes);
    hipMallocManaged((void**)&P, nBytes);

    // 初始化数据
    for (int i = 0; i < width; i++) {
        A[i] = rand() % 100;
    }

    // 初始化mask
    int *h_mask = new int[MASK_WIDTH];
    for (int i = 0; i < MASK_WIDTH; i++) {
        h_mask[i] = rand() % 10;
    }
    // 将数据copy到常量内存
    hipMemcpyToSymbol(HIP_SYMBOL(M), h_mask, mBytes);

    // 每个block需要更多线程读数据
    dim3 blockSize(BLOCK_WIDTH, 1);
    // 每个block需要写O_TILE_WIDTH个数据
    dim3 gridSize((width + O_TILE_WIDTH - 1) / O_TILE_WIDTH, 1);
    // 执行kernel
    tiled1DConvKernel <<< gridSize, blockSize >>>(A, P, width);
    // 等待所有线程执行完毕
    //hipDeviceSynchronize();
    // 检查结果
    /**
    for (int i = 0; i < width; i++) {
        printf("id=%d, P=%d\n", i, P[i]);
    }
    **/
    verify_result(A, P, h_mask, width);
    std::cout << "COMPLETED SUCCESSFULLY\n";

    return 0;
}