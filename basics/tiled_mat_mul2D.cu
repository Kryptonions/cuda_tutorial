#include "hip/hip_runtime.h"
#include <iostream>
#include "../tools/common.cuh"

using namespace std;

const int TILE_WIDTH = 16; // 必须加const
//extern __shared__ float sharedA[][];
//extern __shared__ float sharedB[][];

struct Matrix
{
    int width;
    int height;
    float *elements;
};

// 获取矩阵A的(row, col)元素
__device__ float getElement(Matrix *A, int row, int col)
{
	return A->elements[row * A->width + col];
}

// 为矩阵A的(row, col)元素赋值
__device__ void setElement(Matrix *A, int row, int col, float value)
{
	A->elements[row * A->width + col] = value;
}

// 矩阵相乘kernel，2-D，每个线程计算一个元素
__global__ void tiledMatMulKernel(Matrix *A, Matrix *B, Matrix *C)
{   
    // block内共享内存，矩阵维度必须是常量，否则编译报错
    __shared__ float sharedA[TILE_WIDTH][TILE_WIDTH];  
    __shared__ float sharedB[TILE_WIDTH][TILE_WIDTH];
	  float Cvalue = 0.0;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
	int row = ty + by * blockDim.y;  //线程计算的元素所在的行，(row, col)表示计算元素的矩阵索引
	int col = tx + bx * blockDim.x;  //线程计算的元素所在的列，(row, col)表示计算元素的矩阵索引
    int m = A->height;                //A的行数
    int n = A->width;                 //A的列数
    int k = B->width;                 //B的列数
    for (int i = 0; i < (n - 1) / TILE_WIDTH + 1; i++) {
        //保证A的元素valid
        if(row < m && i * TILE_WIDTH + tx < n) {
            sharedA[ty][tx] = getElement(A, row, i * TILE_WIDTH + tx);
        } else {
            sharedA[ty][tx] = 0.0;
        }
        //保证B的元素valid
        if (col < k && i * TILE_WIDTH + ty < n) {
            sharedB[ty][tx] = getElement(B, i * TILE_WIDTH + ty, col);
        } else {
            sharedA[ty][tx] = 0.0;
        } 
        // 等待block内所有线程读取数据到共享内存
        __syncthreads();
        for (int j = 0; j < TILE_WIDTH; j++) {
            Cvalue += sharedA[ty][j] * sharedB[j][tx];
        }
        // 等待block内所有线程计算得到
        __syncthreads();
    }
	  if (row < m && col < k) {
        setElement(C, row, col, Cvalue);
    }
}

int main(void)
{
    // 1、设置GPU设备
    setGPU();

    // 2、分配主机内存和设备内存，并初始化
    int width = 1 << 10;
    int height = 1 << 10;


    Matrix *A, *B, *C;
    // 申请托管内存
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));
    int nBytes = width * height * sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    // 初始化数据
    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }


    // 定义kernel的执行配置
    dim3 blockSize(TILE_WIDTH, TILE_WIDTH);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
        (height + blockSize.y - 1) / blockSize.y);
    // 执行kernel
    tiledMatMulKernel <<< gridSize, blockSize >>>(A, B, C);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < width * height; ++i)
        maxError = fmax(maxError, fabs(C->elements[i] - 2 * width));
    cout << "最大误差: " << maxError << endl;

    return 0;
}